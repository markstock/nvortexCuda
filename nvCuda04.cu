/*
 * nvortexCuda.cpp
 *
 * (c)2022 Mark J. Stock <markjstock@gmail.com>
 *
 * v0.4  use Kahan summation to improve answer
 */

#include <vector>
#include <random>
#include <chrono>

#include <hip/hip_runtime.h>


// compute using float or double
#define FLOAT2 float2
#define FLOAT float

// threads per block (hard coded)
#define THREADS_PER_BLOCK 256

// GPU count limit
#define MAX_GPUS 8

// Kahan summation
// use single precision for the storage and arithmetic, but accumulation acts close to double-precision
// sum.x is running sum, sum.y is compensation/error
__device__ inline void KahanSum_gpu (const FLOAT toadd, FLOAT2* const sum) {
  const FLOAT y = toadd - (*sum).y;
  const FLOAT t = (*sum).x + y;
  (*sum).y = (t - (*sum).x) - y;
  (*sum).x = t;
}

// -------------------------
// compute kernel - GPU
__global__ void nvortex_2d_nograds_gpu(
    const int32_t nSrc,
    const FLOAT* const __restrict__ sx,
    const FLOAT* const __restrict__ sy,
    const FLOAT* const __restrict__ ss,
    const FLOAT* const __restrict__ sr,
    const int32_t tOffset,
    const FLOAT* const __restrict__ tx,
    const FLOAT* const __restrict__ ty,
    const FLOAT* const __restrict__ tr,
    FLOAT* const __restrict__ tu,
    FLOAT* const __restrict__ tv) {

  // local "thread" id - this is the target particle
  const int32_t i = tOffset + blockIdx.x*THREADS_PER_BLOCK + threadIdx.x;

  // load sources into shared memory (or not)
  __shared__ FLOAT s_sx[THREADS_PER_BLOCK];
  __shared__ FLOAT s_sy[THREADS_PER_BLOCK];
  __shared__ FLOAT s_ss[THREADS_PER_BLOCK];
  __shared__ FLOAT s_sr[THREADS_PER_BLOCK];

  // velocity accumulators for target point
  FLOAT2 locu = make_float2(0.f, 0.f);
  FLOAT2 locv = make_float2(0.f, 0.f);

  FLOAT tr2 = tr[i]*tr[i];

  // which sources do we iterate over?
  const int32_t jcount = nSrc / gridDim.y;
  const int32_t jstart = blockIdx.y * jcount;

  for (int32_t b=0; b<jcount/THREADS_PER_BLOCK; ++b) {
    __syncthreads();

    const int32_t gidx = jstart + b*THREADS_PER_BLOCK + threadIdx.x;
    s_sx[threadIdx.x] = sx[gidx];
    s_sy[threadIdx.x] = sy[gidx];
    s_ss[threadIdx.x] = ss[gidx];
    s_sr[threadIdx.x] = sr[gidx];
    __syncthreads();

    // loop over all source points
    for (int32_t j=0; j<THREADS_PER_BLOCK; ++j) {
      FLOAT dx = s_sx[j] - tx[i];
      FLOAT dy = s_sy[j] - ty[i];
      FLOAT distsq = dx*dx + dy*dy + s_sr[j]*s_sr[j] + tr2;
      // we get __fdividef(x, y) with --use_fast_math it seems
      FLOAT factor = s_ss[j] / distsq;
      KahanSum_gpu( dy * factor, &locu);
      KahanSum_gpu(-dx * factor, &locv);
    }
  }

  // save into device view with atomics
  atomicAdd(&tu[i], (locu.x+locu.y) / (2.0f*3.1415926536f));
  atomicAdd(&tv[i], (locv.x+locv.y) / (2.0f*3.1415926536f));

  return;
}

// Kahan summation
// use single precision for the storage and arithmetic, but accumulation acts close to double-precision
// sum.x is running sum, sum.y is compensation/error
#pragma omp declare simd
__host__ inline void KahanSum_cpu (const FLOAT toadd, FLOAT* const sum, FLOAT* const rem) {
  const FLOAT y = toadd - *rem;
  const FLOAT t = *sum + y;
  *rem = (t - *sum) - y;
  *sum = t;
}

// -------------------------
// compute kernel - CPU
__host__ void nvortex_2d_nograds_cpu(
    const int32_t nSrc,
    const FLOAT* const __restrict__ sx,
    const FLOAT* const __restrict__ sy,
    const FLOAT* const __restrict__ ss,
    const FLOAT* const __restrict__ sr,
    const FLOAT tx,
    const FLOAT ty,
    const FLOAT tr,
    FLOAT* const __restrict__ tu,
    FLOAT* const __restrict__ tv) {

  // velocity accumulators for target point
  FLOAT locu = 0.0f;
  FLOAT locv = 0.0f;
  FLOAT ukah = 0.0f;
  FLOAT vkah = 0.0f;

  const FLOAT tr2 = tr*tr;

  // loop over all source points
  #pragma omp simd reduction(+:locu,locv)
  for (int32_t j=0; j<nSrc; ++j) {
    FLOAT dx = sx[j] - tx;
    FLOAT dy = sy[j] - ty;
    FLOAT distsq = dx*dx + dy*dy + sr[j]*sr[j] + tr2;
    FLOAT factor = ss[j] / distsq;
    // I just can't get this to simd-ize!!!
    //{
      FLOAT y = dy * factor - ukah;
      FLOAT t = locu + y;
      ukah = (t - locu) - y;
      locu = t;
    //}
    //{
      y = -dx * factor - vkah;
      t = locv + y;
      vkah = (t - locv) - y;
      locv = t;
    //}
    //KahanSum_cpu( dy * factor, &locu, &ukah);
    //KahanSum_cpu(-dx * factor, &locv, &vkah);
    //locu += dy * factor;
    //locv -= dx * factor;
  }

  // save into device view
  // use atomics?!?
  *tu = (locu+ukah) / (2.0f*3.1415926536f);
  *tv = (locv+vkah) / (2.0f*3.1415926536f);

  return;
}

// not really alignment, just minimum block sizes
__host__ int32_t buffer(const int32_t _n, const int32_t _align) {
  // 63,64 returns 1; 64,64 returns 1; 65,64 returns 2
  return _align*(1+(_n-1)/_align);
}

// main program

static void usage() {
  fprintf(stderr, "Usage: nvCuda04 [-n=<number>]\n");
  exit(1);
}

int main(int argc, char **argv) {

  // number of particles/points
  int32_t npart = 400000;

  if (argc > 1) {
    if (strncmp(argv[1], "-n=", 3) == 0) {
      int num = atoi(argv[1] + 3);
      if (num < 1) usage();
      npart = num;
    }
  }

  printf( "performing 2D vortex Biot-Savart on %d points\n", npart);

  // number of GPUs present
  int32_t ngpus = 1;
  hipGetDeviceCount(&ngpus);
  //ngpus = 1;	// Force 1 GPU
  // number of cuda streams to break work into
  int32_t nstreams = std::min(MAX_GPUS, ngpus);
  printf( "  ngpus ( %d )  and nstreams ( %d )\n", ngpus, nstreams);

  // we parallelize targets over GPUs/streams
  const int32_t ntargperstrm = buffer(npart/nstreams, THREADS_PER_BLOCK*nstreams);
  const int32_t ntargpad = ntargperstrm * nstreams;
  printf( "  ntargperstrm ( %d )  and ntargpad ( %d )\n", ntargperstrm, ntargpad);

  // and on each GPU, we parallelize over THREADS_PER_BLOCK targets and nsrcblocks source blocks
  // number of blocks source-wise (break summations over sources into this many chunks)
  const int32_t nsrcblocks = 32;

  // set stream sizes
  const int32_t nsrcpad = buffer(npart, THREADS_PER_BLOCK*nsrcblocks);
  const int32_t nsrcperblock = nsrcpad / nsrcblocks;
  printf( "  nsrcperblock ( %d )  and nsrcpad ( %d )\n", nsrcperblock, nsrcpad);

  // define the host arrays (for now, sources and targets are the same)
  const int32_t npad = std::max(ntargpad,nsrcpad);
  std::vector<FLOAT> hsx(npad), hsy(npad), hss(npad), hsr(npad), htu(npad), htv(npad);
  const FLOAT thisstrmag = 1.0 / std::sqrt(npart);
  const FLOAT thisrad    = (2./3.) / std::sqrt(npart);
  //std::random_device dev;
  //std::mt19937 rng(dev());
  std::mt19937 rng(1234);
  std::uniform_real_distribution<FLOAT> xrand(0.0,1.0);
  for (int32_t i = 0; i < npart; ++i)    hsx[i] = xrand(rng);
  for (int32_t i = npart; i < npad; ++i) hsx[i] = 0.0;
  for (int32_t i = 0; i < npart; ++i)    hsy[i] = xrand(rng);
  for (int32_t i = npart; i < npad; ++i) hsy[i] = 0.0;
  for (int32_t i = 0; i < npart; ++i)    hss[i] = thisstrmag * (2.0*xrand(rng)-1.0);
  for (int32_t i = npart; i < npad; ++i) hss[i] = 0.0;
  for (int32_t i = 0; i < npart; ++i)    hsr[i] = thisrad;
  for (int32_t i = npart; i < npad; ++i) hsr[i] = thisrad;
  for (int32_t i = 0; i < npad; ++i)     htu[i] = 0.0;
  for (int32_t i = 0; i < npad; ++i)     htv[i] = 0.0;

  // -------------------------
  // do a CPU version

  auto start = std::chrono::system_clock::now();

  #pragma omp parallel for
  for (int32_t i=0; i<npart; ++i) {
    nvortex_2d_nograds_cpu(npart, hsx.data(),hsy.data(),hss.data(),hsr.data(), hsx[i],hsy[i],hsr[i], &htu[i],&htv[i]);
  }

  auto end = std::chrono::system_clock::now();
  std::chrono::duration<double> elapsed_seconds = end-start;
  double time = elapsed_seconds.count();

  printf( "  host total time( %g s ) and flops( %g GFlop/s )\n", time, 1.e-9 * (double)npart*(5+19*(double)npart)/time);
  printf( "    results ( %g %g %g %g %g %g)\n", htu[0], htv[0], htu[1], htv[1], htu[npart-1], htv[npart-1]);

  // copy the results into temp vectors
  std::vector<FLOAT> htu_cpu(htu);
  std::vector<FLOAT> htv_cpu(htv);

  // -------------------------
  // do the GPU version

  // set device pointers, too
  FLOAT *dsx[MAX_GPUS], *dsy[MAX_GPUS], *dss[MAX_GPUS], *dsr[MAX_GPUS];
  FLOAT *dtx[MAX_GPUS], *dty[MAX_GPUS], *dtr[MAX_GPUS];
  FLOAT *dtu[MAX_GPUS], *dtv[MAX_GPUS];
  hipStream_t stream[MAX_GPUS];

  // allocate space for all sources, part of targets
  const int32_t srcsize = nsrcpad*sizeof(FLOAT);
  const int32_t trgsize = ntargperstrm*sizeof(FLOAT);
  for (int32_t i=0; i<nstreams; ++i) {
    hipSetDevice(i);
    hipStreamCreate(&stream[i]);

    hipMalloc (&dsx[i], srcsize);
    hipMalloc (&dsy[i], srcsize);
    hipMalloc (&dss[i], srcsize);
    hipMalloc (&dsr[i], srcsize);
    hipMalloc (&dtu[i], trgsize);
    hipMalloc (&dtv[i], trgsize);
  }

  // to be fair, we start timer after allocation but before transfer
  start = std::chrono::system_clock::now();

  // now perform the data movement and setting
  for (int32_t i=0; i<nstreams; ++i) {

    hipSetDevice(i);

    // move the data
    hipMemcpyAsync (dsx[i], hsx.data(), srcsize, hipMemcpyHostToDevice, stream[i]);
    hipMemcpyAsync (dsy[i], hsy.data(), srcsize, hipMemcpyHostToDevice, stream[i]);
    hipMemcpyAsync (dss[i], hss.data(), srcsize, hipMemcpyHostToDevice, stream[i]);
    hipMemcpyAsync (dsr[i], hsr.data(), srcsize, hipMemcpyHostToDevice, stream[i]);
    hipMemsetAsync (dtu[i], 0, trgsize, stream[i]);
    hipMemsetAsync (dtv[i], 0, trgsize, stream[i]);
    // now we need to be careful to point to the part of the source arrays that hold
    //   just this GPUs set of target particles
    dtx[i] = dsx[i] + i*ntargperstrm;
    dty[i] = dsy[i] + i*ntargperstrm;
    dtr[i] = dsr[i] + i*ntargperstrm;

    // check
    auto memerr = hipGetLastError();
    if (memerr != hipSuccess) {
      fprintf(stderr, "Failed to upload data (other): %s!\n", hipGetErrorString(memerr));
      exit(EXIT_FAILURE);
    }
  }

  const dim3 blocksz(THREADS_PER_BLOCK, 1, 1);
  const dim3 gridsz(ntargperstrm/THREADS_PER_BLOCK, nsrcblocks, 1);

  for (int32_t i=0; i<nstreams; ++i) {
    // launch the kernel
    hipSetDevice(i);
    nvortex_2d_nograds_gpu<<<gridsz,blocksz,0,stream[i]>>>(nsrcpad, dsx[i],dsy[i],dss[i],dsr[i],
                                               0,dtx[i],dty[i],dtr[i],dtu[i],dtv[i]);

    // check
    auto err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Failed to launch kernel (%d): %s!\n", i, hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
  }

  for (int32_t i=0; i<nstreams; ++i) {
    // pull data back down
    hipMemcpyAsync (htu.data() + i*ntargperstrm, dtu[i], trgsize, hipMemcpyDeviceToHost, stream[i]);
    hipMemcpyAsync (htv.data() + i*ntargperstrm, dtv[i], trgsize, hipMemcpyDeviceToHost, stream[i]);
  }

  // join streams
  for (int32_t i=0; i<nstreams; ++i) {
    hipStreamSynchronize(stream[i]);
  }
  //cudaDeviceSynchronize();

  // time and report
  end = std::chrono::system_clock::now();
  elapsed_seconds = end-start;
  time = elapsed_seconds.count();
  printf( "  device total time( %g s ) and flops( %g GFlop/s )\n", time, 1.e-9 * (double)npart*(5+19*(double)npart)/time);
  printf( "    results ( %g %g %g %g %g %g)\n", htu[0], htv[0], htu[1], htv[1], htu[npart-1], htv[npart-1]);

  // free resources, after timer
  for (int32_t i=0; i<nstreams; ++i) {
    hipFree(dsx[i]);
    hipFree(dsy[i]);
    hipFree(dss[i]);
    hipFree(dsr[i]);
    hipFree(dtu[i]);
    hipFree(dtv[i]);
    hipStreamDestroy(stream[i]);
  }

  // compare results
  FLOAT errsum = 0.0;
  FLOAT errmax = 0.0;
  for (int32_t i=0; i<npart; ++i) {
    const FLOAT thiserr = std::pow(htu[i]-htu_cpu[i], 2) + std::pow(htv[i]-htv_cpu[i], 2);
    errsum += thiserr;
    if ((FLOAT)std::sqrt(thiserr) > errmax) {
      errmax = (FLOAT)std::sqrt(thiserr);
      //printf( "    err at %d is %g\n", i, errmax);
    }
  }
  printf( "  total host-device error ( %g ) max error ( %g )\n", std::sqrt(errsum/npart), errmax);
}

