/*
 * ngCuda03.cpp
 *
 * (c)2022 Mark J. Stock <markjstock@gmail.com>
 *
 * v0.3  use atomics to expose more concurrency
 */

#include <vector>
#include <random>
#include <chrono>

#include <hip/hip_runtime.h>


// compute using float or double
#define FLOAT float

// threads per block (hard coded)
#define THREADS_PER_BLOCK 128

// GPU count limit
#define MAX_GPUS 8

// -------------------------
// compute kernel - GPU
__global__ void ngrav_3d_nograds_gpu(
    const int32_t nSrc,
    const FLOAT* const __restrict__ sx,
    const FLOAT* const __restrict__ sy,
    const FLOAT* const __restrict__ sz,
    const FLOAT* const __restrict__ ss,
    const FLOAT* const __restrict__ sr,
    const int32_t tOffset,
    const FLOAT* const __restrict__ tx,
    const FLOAT* const __restrict__ ty,
    const FLOAT* const __restrict__ tz,
    const FLOAT* const __restrict__ tr,
    FLOAT* const __restrict__ tu,
    FLOAT* const __restrict__ tv,
    FLOAT* const __restrict__ tw) {

  // local "thread" id - this is the target particle
  const int32_t i = tOffset + blockIdx.x*THREADS_PER_BLOCK + threadIdx.x;

  // load sources into shared memory (or not)
  __shared__ FLOAT s_sx[THREADS_PER_BLOCK];
  __shared__ FLOAT s_sy[THREADS_PER_BLOCK];
  __shared__ FLOAT s_sz[THREADS_PER_BLOCK];
  __shared__ FLOAT s_ss[THREADS_PER_BLOCK];
  __shared__ FLOAT s_sr[THREADS_PER_BLOCK];

  // velocity accumulators for target point
  FLOAT locu = 0.0f;
  FLOAT locv = 0.0f;
  FLOAT locw = 0.0f;

  // which sources do we iterate over?
  const int32_t jcount = nSrc / gridDim.y;
  const int32_t jstart = blockIdx.y * jcount;

  for (int32_t b=0; b<jcount/THREADS_PER_BLOCK; ++b) {
    __syncthreads();

    const int32_t gidx = jstart + b*THREADS_PER_BLOCK + threadIdx.x;
    s_sx[threadIdx.x] = sx[gidx];
    s_sy[threadIdx.x] = sy[gidx];
    s_sz[threadIdx.x] = sz[gidx];
    s_ss[threadIdx.x] = ss[gidx];
    s_sr[threadIdx.x] = sr[gidx];
    __syncthreads();

    // loop over all source points
    for (int32_t j=0; j<THREADS_PER_BLOCK; ++j) {
      FLOAT dx = s_sx[j] - tx[i];
      FLOAT dy = s_sy[j] - ty[i];
      FLOAT dz = s_sz[j] - tz[i];
      FLOAT distsq = dx*dx + dy*dy + dz*dz + s_sr[j]*s_sr[j] + tr[i]*tr[i];
      FLOAT invR = rsqrtf(distsq);
      FLOAT invR2 = invR*invR;
      FLOAT factor = s_ss[j] * invR * invR2;
      locu += dx * factor;
      locv += dy * factor;
      locw += dz * factor;
    }
  }

  // save into device view
  // use atomics
  atomicAdd(&tu[i], locu / (4.0f*3.1415926536f));
  atomicAdd(&tv[i], locv / (4.0f*3.1415926536f));
  atomicAdd(&tw[i], locw / (4.0f*3.1415926536f));

  return;
}

// -------------------------
// compute kernel - CPU
__host__ void ngrav_3d_nograds_cpu(
    const int32_t nSrc,
    const FLOAT* const __restrict__ sx,
    const FLOAT* const __restrict__ sy,
    const FLOAT* const __restrict__ sz,
    const FLOAT* const __restrict__ ss,
    const FLOAT* const __restrict__ sr,
    const FLOAT tx,
    const FLOAT ty,
    const FLOAT tz,
    const FLOAT tr,
    FLOAT* const __restrict__ tu,
    FLOAT* const __restrict__ tv,
    FLOAT* const __restrict__ tw) {

  // velocity accumulators for target point
  FLOAT locu = 0.0f;
  FLOAT locv = 0.0f;
  FLOAT locw = 0.0f;

  // loop over all source points
  #pragma omp simd reduction(+:locu,locv)
  for (int32_t j=0; j<nSrc; ++j) {
    FLOAT dx = sx[j] - tx;
    FLOAT dy = sy[j] - ty;
    FLOAT dz = sz[j] - tz;
    FLOAT distsq = dx*dx + dy*dy + dz*dz + sr[j]*sr[j] + tr*tr;
    FLOAT invR = rsqrt(distsq);
    FLOAT invR2 = invR*invR;
    FLOAT factor = ss[j] * invR * invR2;
    locu += dx * factor;
    locv += dy * factor;
    locw += dz * factor;
  }

  // save into device view
  // use atomics?!?
  *tu = locu / (4.0f*3.1415926536f);
  *tv = locv / (4.0f*3.1415926536f);
  *tw = locw / (4.0f*3.1415926536f);

  return;
}

// not really alignment, just minimum block sizes
__host__ int32_t buffer(const int32_t _n, const int32_t _align) {
  // 63,64 returns 1; 64,64 returns 1; 65,64 returns 2
  return _align*(1+(_n-1)/_align);
}

// main program

static void usage() {
  fprintf(stderr, "Usage: ngCuda03 [-n=<number>]\n");
  exit(1);
}

int main(int argc, char **argv) {

  // number of particles/points
  int32_t npart = 400000;

  if (argc > 1) {
    if (strncmp(argv[1], "-n=", 3) == 0) {
      int num = atoi(argv[1] + 3);
      if (num < 1) usage();
      npart = num;
    }
  }

  printf( "performing 3D gravitational direct summation on %d points\n", npart);

  // number of GPUs present
  int32_t ngpus = 1;
  hipGetDeviceCount(&ngpus);
  //ngpus = 1;	// Force 1 GPU
  // number of cuda streams to break work into
  int32_t nstreams = std::min(MAX_GPUS, ngpus);
  printf( "  ngpus ( %d )  and nstreams ( %d )\n", ngpus, nstreams);

  // we parallelize targets over GPUs/streams
  const int32_t ntargperstrm = buffer(npart/nstreams, THREADS_PER_BLOCK*nstreams);
  const int32_t ntargpad = ntargperstrm * nstreams;
  printf( "  ntargperstrm ( %d )  and ntargpad ( %d )\n", ntargperstrm, ntargpad);

  // and on each GPU, we parallelize over THREADS_PER_BLOCK targets and nsrcblocks source blocks
  // number of blocks source-wise (break summations over sources into this many chunks)
  const int32_t nsrcblocks = 32;

  // set stream sizes
  const int32_t nsrcpad = buffer(npart, THREADS_PER_BLOCK*nsrcblocks);
  const int32_t nsrcperblock = nsrcpad / nsrcblocks;
  printf( "  nsrcperblock ( %d )  and nsrcpad ( %d )\n", nsrcperblock, nsrcpad);

  // define the host arrays (for now, sources and targets are the same)
  const int32_t npad = std::max(ntargpad,nsrcpad);
  std::vector<FLOAT> hsx(npad), hsy(npad), hsz(npad), hss(npad), hsr(npad), htu(npad), htv(npad), htw(npad);
  const FLOAT thisstrmag = 1.0 / std::sqrt(npart);
  const FLOAT thisrad    = (2./3.) / std::sqrt(npart);
  //std::random_device dev;
  //std::mt19937 rng(dev());
  std::mt19937 rng(1234);
  std::uniform_real_distribution<FLOAT> xrand(0.0,1.0);
  for (int32_t i = 0; i < npart; ++i)    hsx[i] = xrand(rng);
  for (int32_t i = npart; i < npad; ++i) hsx[i] = 0.0;
  for (int32_t i = 0; i < npart; ++i)    hsy[i] = xrand(rng);
  for (int32_t i = npart; i < npad; ++i) hsy[i] = 0.0;
  for (int32_t i = 0; i < npart; ++i)    hsz[i] = xrand(rng);
  for (int32_t i = npart; i < npad; ++i) hsz[i] = 0.0;
  for (int32_t i = 0; i < npart; ++i)    hss[i] = thisstrmag * (2.0*xrand(rng)-1.0);
  for (int32_t i = npart; i < npad; ++i) hss[i] = 0.0;
  for (int32_t i = 0; i < npart; ++i)    hsr[i] = thisrad;
  for (int32_t i = npart; i < npad; ++i) hsr[i] = thisrad;
  for (int32_t i = 0; i < npad; ++i)     htu[i] = 0.0;
  for (int32_t i = 0; i < npad; ++i)     htv[i] = 0.0;
  for (int32_t i = 0; i < npad; ++i)     htw[i] = 0.0;

  // -------------------------
  // do a CPU version

  auto start = std::chrono::system_clock::now();

  #pragma omp parallel for
  for (int32_t i=0; i<npart; ++i) {
    ngrav_3d_nograds_cpu(npart, hsx.data(),hsy.data(),hsz.data(),hss.data(),hsr.data(), hsx[i],hsy[i],hsz[i],hsr[i], &htu[i],&htv[i],&htw[i]);
  }

  auto end = std::chrono::system_clock::now();
  std::chrono::duration<double> elapsed_seconds = end-start;
  double time = elapsed_seconds.count();

  printf( "  host total time( %g s ) and flops( %g GFlop/s )\n", time, 1.e-9 * (double)npart*(6+22*(double)npart)/time);
  for (int i=0; i<4; ++i) {
    printf( "    part %d acc %g %g %g)\n", i, htu[i], htv[i], htw[i]);
  }

  // copy the results into temp vectors
  std::vector<FLOAT> htu_cpu(htu);
  std::vector<FLOAT> htv_cpu(htv);
  std::vector<FLOAT> htw_cpu(htw);

  // -------------------------
  // do the GPU version

  // set device pointers, too
  FLOAT *dsx[MAX_GPUS], *dsy[MAX_GPUS], *dsz[MAX_GPUS], *dss[MAX_GPUS], *dsr[MAX_GPUS];
  FLOAT *dtx[MAX_GPUS], *dty[MAX_GPUS], *dtz[MAX_GPUS], *dtr[MAX_GPUS];
  FLOAT *dtu[MAX_GPUS], *dtv[MAX_GPUS], *dtw[MAX_GPUS];
  hipStream_t stream[MAX_GPUS];

  // allocate space for all sources, part of targets
  const int32_t srcsize = nsrcpad*sizeof(FLOAT);
  const int32_t trgsize = ntargperstrm*sizeof(FLOAT);
  for (int32_t i=0; i<nstreams; ++i) {
    hipSetDevice(i);
    hipStreamCreate(&stream[i]);

    hipMalloc (&dsx[i], srcsize);
    hipMalloc (&dsy[i], srcsize);
    hipMalloc (&dsz[i], srcsize);
    hipMalloc (&dss[i], srcsize);
    hipMalloc (&dsr[i], srcsize);
    hipMalloc (&dtu[i], trgsize);
    hipMalloc (&dtv[i], trgsize);
    hipMalloc (&dtw[i], trgsize);
  }

  // to be fair, we start timer after allocation but before transfer
  start = std::chrono::system_clock::now();

  // now perform the data movement and setting
  for (int32_t i=0; i<nstreams; ++i) {

    hipSetDevice(i);

    // move the data
    hipMemcpyAsync (dsx[i], hsx.data(), srcsize, hipMemcpyHostToDevice, stream[i]);
    hipMemcpyAsync (dsy[i], hsy.data(), srcsize, hipMemcpyHostToDevice, stream[i]);
    hipMemcpyAsync (dsz[i], hsz.data(), srcsize, hipMemcpyHostToDevice, stream[i]);
    hipMemcpyAsync (dss[i], hss.data(), srcsize, hipMemcpyHostToDevice, stream[i]);
    hipMemcpyAsync (dsr[i], hsr.data(), srcsize, hipMemcpyHostToDevice, stream[i]);
    hipMemsetAsync (dtu[i], 0, trgsize, stream[i]);
    hipMemsetAsync (dtv[i], 0, trgsize, stream[i]);
    hipMemsetAsync (dtw[i], 0, trgsize, stream[i]);
    // now we need to be careful to point to the part of the source arrays that hold
    //   just this GPUs set of target particles
    dtx[i] = dsx[i] + i*ntargperstrm;
    dty[i] = dsy[i] + i*ntargperstrm;
    dtz[i] = dsz[i] + i*ntargperstrm;
    dtr[i] = dsr[i] + i*ntargperstrm;

    // check
    auto memerr = hipGetLastError();
    if (memerr != hipSuccess) {
      fprintf(stderr, "Failed to upload data (other): %s!\n", hipGetErrorString(memerr));
      exit(EXIT_FAILURE);
    }
  }

    const dim3 blocksz(THREADS_PER_BLOCK, 1, 1);
    const dim3 gridsz(ntargperstrm/THREADS_PER_BLOCK, nsrcblocks, 1);

  for (int32_t i=0; i<nstreams; ++i) {
    // launch the kernel
    hipSetDevice(i);
    ngrav_3d_nograds_gpu<<<gridsz,blocksz,0,stream[i]>>>(nsrcpad, dsx[i],dsy[i],dsz[i],dss[i],dsr[i],
                                               0,dtx[i],dty[i],dtz[i],dtr[i],dtu[i],dtv[i],dtw[i]);

    // check
    auto err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Failed to launch kernel (%d): %s!\n", i, hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
  }

  for (int32_t i=0; i<nstreams; ++i) {
    // pull data back down
    hipMemcpyAsync (htu.data() + i*ntargperstrm, dtu[i], trgsize, hipMemcpyDeviceToHost, stream[i]);
    hipMemcpyAsync (htv.data() + i*ntargperstrm, dtv[i], trgsize, hipMemcpyDeviceToHost, stream[i]);
    hipMemcpyAsync (htw.data() + i*ntargperstrm, dtw[i], trgsize, hipMemcpyDeviceToHost, stream[i]);
  }

  // join streams
  for (int32_t i=0; i<nstreams; ++i) {
    hipStreamSynchronize(stream[i]);
  }
  //cudaDeviceSynchronize();

  // time and report
  end = std::chrono::system_clock::now();
  elapsed_seconds = end-start;
  time = elapsed_seconds.count();
  printf( "  device total time( %g s ) and flops( %g GFlop/s )\n", time, 1.e-9 * (double)npart*(6+22*(double)npart)/time);
  for (int i=0; i<4; ++i) {
    printf( "    part %d acc %g %g %g)\n", i, htu[i], htv[i], htw[i]);
  }

  // free resources, after timer
  for (int32_t i=0; i<nstreams; ++i) {
    hipFree(dsx[i]);
    hipFree(dsy[i]);
    hipFree(dsz[i]);
    hipFree(dss[i]);
    hipFree(dsr[i]);
    hipFree(dtu[i]);
    hipFree(dtv[i]);
    hipFree(dtw[i]);
    hipStreamDestroy(stream[i]);
  }

  // compare results
  FLOAT errsum = 0.0;
  FLOAT errmax = 0.0;
  for (int32_t i=0; i<npart; ++i) {
    const FLOAT thiserr = std::pow(htu[i]-htu_cpu[i], 2) + std::pow(htv[i]-htv_cpu[i], 2);
    errsum += thiserr;
    if ((FLOAT)std::sqrt(thiserr) > errmax) {
      errmax = (FLOAT)std::sqrt(thiserr);
      //printf( "    err at %d is %g\n", i, errmax);
    }
  }
  printf( "  total host-device error ( %g ) max error ( %g )\n", std::sqrt(errsum/npart), errmax);
}

