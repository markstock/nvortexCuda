/*
 * ngCuda03.cpp
 *
 * (c)2022 Mark J. Stock <markjstock@gmail.com>
 *
 * v0.3  use atomics to expose more concurrency
 * v0.4  use float4 to effectively unroll loops
 */

#include <vector>
#include <random>
#include <chrono>

#include <hip/hip_runtime.h>


// compute using float or double
#define FLOAT float
#define RSQRT rsqrtf

// threads per block (hard coded)
#define THREADS_PER_BLOCK 128

// GPU count limit
#define MAX_GPUS 8

// -------------------------
// compute kernel - GPU
__global__ void ngrav_3d_nograds_gpu(
    const int32_t nSrc,
    const FLOAT* const __restrict__ sx,
    const FLOAT* const __restrict__ sy,
    const FLOAT* const __restrict__ sz,
    const FLOAT* const __restrict__ ss,
    const FLOAT* const __restrict__ sr,
    const int32_t tOffset,
    const FLOAT* const __restrict__ tx,
    const FLOAT* const __restrict__ ty,
    const FLOAT* const __restrict__ tz,
    const FLOAT* const __restrict__ tr,
    FLOAT* const __restrict__ tu,
    FLOAT* const __restrict__ tv,
    FLOAT* const __restrict__ tw) {

  // local "thread" id - this is the target particle
  const int32_t i = tOffset + blockIdx.x*THREADS_PER_BLOCK + threadIdx.x;

  // load sources into shared memory (or not)
  __shared__ FLOAT s_sx[THREADS_PER_BLOCK];
  __shared__ FLOAT s_sy[THREADS_PER_BLOCK];
  __shared__ FLOAT s_sz[THREADS_PER_BLOCK];
  __shared__ FLOAT s_ss[THREADS_PER_BLOCK];
  __shared__ FLOAT s_sr[THREADS_PER_BLOCK];

  // velocity accumulators for target point
  float4 locu = make_float4(0.0f,0.0f,0.0f,0.0f);
  float4 locv = make_float4(0.0f,0.0f,0.0f,0.0f);
  float4 locw = make_float4(0.0f,0.0f,0.0f,0.0f);

  const float tr2 = tr[i]*tr[i];

  // which sources do we iterate over?
  const int32_t jcount = nSrc / gridDim.y;
  const int32_t jstart = blockIdx.y * jcount;

  for (int32_t b=0; b<jcount/THREADS_PER_BLOCK; ++b) {

    const int32_t gidx = jstart + b*THREADS_PER_BLOCK + threadIdx.x;
    s_sx[threadIdx.x] = sx[gidx];
    s_sy[threadIdx.x] = sy[gidx];
    s_sz[threadIdx.x] = sz[gidx];
    s_ss[threadIdx.x] = ss[gidx];
    s_sr[threadIdx.x] = sr[gidx];
    __syncthreads();

    // loop over all source points
    for (int32_t j=0; j<THREADS_PER_BLOCK; j+=4) {
      const int32_t jp1 = j+1;
      const int32_t jp2 = j+2;
      const int32_t jp3 = j+3;
      const float4 dx = make_float4(s_sx[j]-tx[i],s_sx[jp1]-tx[i],s_sx[jp2]-tx[i],s_sx[jp3]-tx[i]);
      const float4 dy = make_float4(s_sy[j]-ty[i],s_sy[jp1]-ty[i],s_sy[jp2]-ty[i],s_sy[jp3]-ty[i]);
      const float4 dz = make_float4(s_sz[j]-tz[i],s_sz[jp1]-tz[i],s_sz[jp2]-tz[i],s_sz[jp3]-tz[i]);
      const float4 vr = make_float4(s_sr[j],s_sr[jp1],s_sr[jp2],s_sr[jp3]);
      const float distsqx = dx.x*dx.x + dy.x*dy.x + dz.x*dz.x + vr.x*vr.x + tr2;
      const float distsqy = dx.y*dx.y + dy.y*dy.y + dz.y*dz.y + vr.y*vr.y + tr2;
      const float distsqz = dx.z*dx.z + dy.z*dy.z + dz.z*dz.z + vr.z*vr.z + tr2;
      const float distsqw = dx.w*dx.w + dy.w*dy.w + dz.w*dz.w + vr.w*vr.w + tr2;
      // this extra flop improves time by >10%
      const float4 invR = make_float4(RSQRT(distsqx), RSQRT(distsqy), RSQRT(distsqz), RSQRT(distsqw));
      const float4 invR2 = make_float4(invR.x*invR.x, invR.y*invR.y, invR.z*invR.z, invR.w*invR.w);
      const float factorx = s_ss[j] * invR.x * invR2.x;
      const float factory = s_ss[jp1] * invR.y * invR2.y;
      const float factorz = s_ss[jp2] * invR.z * invR2.z;
      const float factorw = s_ss[jp3] * invR.w * invR2.w;
      //FLOAT factor = s_ss[j] * RSQRT(distsq) / distsq;
      locu.x += dx.x * factorx;
      locu.y += dx.y * factory;
      locu.z += dx.z * factorz;
      locu.w += dx.w * factorw;
      locv.x += dy.x * factorx;
      locv.y += dy.y * factory;
      locv.z += dy.z * factorz;
      locv.w += dy.w * factorw;
      locw.x += dz.x * factorx;
      locw.y += dz.y * factory;
      locw.z += dz.z * factorz;
      locw.w += dz.w * factorw;
    }
    __syncthreads();
  }

  // save into device view
  // use atomics
  atomicAdd(&tu[i], (locu.x+locu.y+locu.z+locu.w) / (4.0f*3.1415926536f));
  atomicAdd(&tv[i], (locv.x+locv.y+locv.z+locv.w) / (4.0f*3.1415926536f));
  atomicAdd(&tw[i], (locw.x+locw.y+locw.z+locw.w) / (4.0f*3.1415926536f));

  return;
}

// -------------------------
// compute kernel - CPU
__host__ void ngrav_3d_nograds_cpu(
    const int32_t nSrc,
    const FLOAT* const __restrict__ sx,
    const FLOAT* const __restrict__ sy,
    const FLOAT* const __restrict__ sz,
    const FLOAT* const __restrict__ ss,
    const FLOAT* const __restrict__ sr,
    const FLOAT tx,
    const FLOAT ty,
    const FLOAT tz,
    const FLOAT tr,
    FLOAT* const __restrict__ tu,
    FLOAT* const __restrict__ tv,
    FLOAT* const __restrict__ tw) {

  // velocity accumulators for target point
  FLOAT locu = 0.0f;
  FLOAT locv = 0.0f;
  FLOAT locw = 0.0f;

  // loop over all source points
  #pragma omp simd reduction(+:locu,locv)
  for (int32_t j=0; j<nSrc; ++j) {
    FLOAT dx = sx[j] - tx;
    FLOAT dy = sy[j] - ty;
    FLOAT dz = sz[j] - tz;
    FLOAT distsq = dx*dx + dy*dy + dz*dz + sr[j]*sr[j] + tr*tr;
    FLOAT invR = rsqrt(distsq);
    FLOAT invR2 = invR*invR;
    FLOAT factor = ss[j] * invR * invR2;
    locu += dx * factor;
    locv += dy * factor;
    locw += dz * factor;
  }

  // save into device view
  // use atomics?!?
  *tu = locu / (4.0f*3.1415926536f);
  *tv = locv / (4.0f*3.1415926536f);
  *tw = locw / (4.0f*3.1415926536f);

  return;
}

// not really alignment, just minimum block sizes
__host__ int32_t buffer(const int32_t _n, const int32_t _align) {
  // 63,64 returns 1; 64,64 returns 1; 65,64 returns 2
  return _align*(1+(_n-1)/_align);
}

// main program

static void usage() {
  fprintf(stderr, "Usage: ngCuda03 [-n=<number>]\n");
  exit(1);
}

int main(int argc, char **argv) {

  // number of particles/points and gpus
  int32_t npart = 400000;
  int32_t force_ngpus = -1;
  bool compare = false;

  for (int i=1; i<argc; i++) {
    if (strncmp(argv[i], "-n=", 3) == 0) {
      int32_t num = atoi(argv[i]+3);
      if (num < 1) usage();
      npart = num;
    } else if (strncmp(argv[i], "-g=", 3) == 0) {
      int32_t num = atof(argv[i]+3);
      if (num < 1 or num > MAX_GPUS) usage();
      force_ngpus = num;
    } else if (strncmp(argv[i], "-c", 2) == 0) {
      compare = true;
    }
  }

  printf( "performing 3D gravitational direct summation on %d points\n", npart);

  // number of GPUs present
  int32_t ngpus = 1;
  hipGetDeviceCount(&ngpus);
  if (force_ngpus > 0) ngpus = force_ngpus;
  // number of cuda streams to break work into
  int32_t nstreams = std::min(MAX_GPUS, ngpus);
  printf( "  ngpus ( %d )  and nstreams ( %d )\n", ngpus, nstreams);

  // we parallelize targets over GPUs/streams
  const int32_t ntargperstrm = buffer(npart/nstreams, THREADS_PER_BLOCK*nstreams);
  const int32_t ntargpad = ntargperstrm * nstreams;
  printf( "  ntargperstrm ( %d )  and ntargpad ( %d )\n", ntargperstrm, ntargpad);

  // and on each GPU, we parallelize over THREADS_PER_BLOCK targets and nsrcblocks source blocks
  // number of blocks source-wise (break summations over sources into this many chunks)
  const int32_t nsrcblocks = 32;

  // set stream sizes
  const int32_t nsrcpad = buffer(npart, THREADS_PER_BLOCK*nsrcblocks);
  const int32_t nsrcperblock = nsrcpad / nsrcblocks;
  printf( "  nsrcperblock ( %d )  and nsrcpad ( %d )\n", nsrcperblock, nsrcpad);

  // define the host arrays (for now, sources and targets are the same)
  const int32_t npad = std::max(ntargpad,nsrcpad);
  std::vector<FLOAT> hsx(npad), hsy(npad), hsz(npad), hss(npad), hsr(npad), htu(npad), htv(npad), htw(npad);
  const FLOAT thisstrmag = 1.0 / std::sqrt(npart);
  const FLOAT thisrad    = (2./3.) / std::sqrt(npart);
  //std::random_device dev;
  //std::mt19937 rng(dev());
  std::mt19937 rng(1234);
  std::uniform_real_distribution<FLOAT> xrand(0.0,1.0);
  for (int32_t i = 0; i < npart; ++i)    hsx[i] = xrand(rng);
  for (int32_t i = npart; i < npad; ++i) hsx[i] = 0.0;
  for (int32_t i = 0; i < npart; ++i)    hsy[i] = xrand(rng);
  for (int32_t i = npart; i < npad; ++i) hsy[i] = 0.0;
  for (int32_t i = 0; i < npart; ++i)    hsz[i] = xrand(rng);
  for (int32_t i = npart; i < npad; ++i) hsz[i] = 0.0;
  for (int32_t i = 0; i < npart; ++i)    hss[i] = thisstrmag * (2.0*xrand(rng)-1.0);
  for (int32_t i = npart; i < npad; ++i) hss[i] = 0.0;
  for (int32_t i = 0; i < npart; ++i)    hsr[i] = thisrad;
  for (int32_t i = npart; i < npad; ++i) hsr[i] = thisrad;
  for (int32_t i = 0; i < npad; ++i)     htu[i] = 0.0;
  for (int32_t i = 0; i < npad; ++i)     htv[i] = 0.0;
  for (int32_t i = 0; i < npad; ++i)     htw[i] = 0.0;

  // -------------------------
  // do a CPU version

  if (compare) {
  auto start = std::chrono::system_clock::now();

  #pragma omp parallel for
  for (int32_t i=0; i<npart; ++i) {
    ngrav_3d_nograds_cpu(npart, hsx.data(),hsy.data(),hsz.data(),hss.data(),hsr.data(), hsx[i],hsy[i],hsz[i],hsr[i], &htu[i],&htv[i],&htw[i]);
  }

  auto end = std::chrono::system_clock::now();
  std::chrono::duration<double> elapsed_seconds = end-start;
  double time = elapsed_seconds.count();

  printf( "  host total time( %g s ) and flops( %g GFlop/s )\n", time, 1.e-9 * (double)npart*(6+21*(double)npart)/time);
  for (int i=0; i<4; ++i) {
    printf( "    part %d acc %g %g %g)\n", i, htu[i], htv[i], htw[i]);
  }
  }

  // copy the results into temp vectors
  std::vector<FLOAT> htu_cpu(htu);
  std::vector<FLOAT> htv_cpu(htv);
  std::vector<FLOAT> htw_cpu(htw);

  // -------------------------
  // do the GPU version

  // set device pointers, too
  FLOAT *dsx[MAX_GPUS], *dsy[MAX_GPUS], *dsz[MAX_GPUS], *dss[MAX_GPUS], *dsr[MAX_GPUS];
  FLOAT *dtx[MAX_GPUS], *dty[MAX_GPUS], *dtz[MAX_GPUS], *dtr[MAX_GPUS];
  FLOAT *dtu[MAX_GPUS], *dtv[MAX_GPUS], *dtw[MAX_GPUS];
  hipStream_t stream[MAX_GPUS];

  // allocate space for all sources, part of targets
  const int32_t srcsize = nsrcpad*sizeof(FLOAT);
  const int32_t trgsize = ntargperstrm*sizeof(FLOAT);
  for (int32_t i=0; i<nstreams; ++i) {
    hipSetDevice(i);
    hipStreamCreate(&stream[i]);

    hipMalloc (&dsx[i], srcsize);
    hipMalloc (&dsy[i], srcsize);
    hipMalloc (&dsz[i], srcsize);
    hipMalloc (&dss[i], srcsize);
    hipMalloc (&dsr[i], srcsize);
    hipMalloc (&dtu[i], trgsize);
    hipMalloc (&dtv[i], trgsize);
    hipMalloc (&dtw[i], trgsize);
  }

  // to be fair, we start timer after allocation but before transfer
  auto start = std::chrono::system_clock::now();

  // now perform the data movement and setting
  for (int32_t i=0; i<nstreams; ++i) {

    hipSetDevice(i);

    // move the data
    hipMemcpyAsync (dsx[i], hsx.data(), srcsize, hipMemcpyHostToDevice, stream[i]);
    hipMemcpyAsync (dsy[i], hsy.data(), srcsize, hipMemcpyHostToDevice, stream[i]);
    hipMemcpyAsync (dsz[i], hsz.data(), srcsize, hipMemcpyHostToDevice, stream[i]);
    hipMemcpyAsync (dss[i], hss.data(), srcsize, hipMemcpyHostToDevice, stream[i]);
    hipMemcpyAsync (dsr[i], hsr.data(), srcsize, hipMemcpyHostToDevice, stream[i]);
    hipMemsetAsync (dtu[i], 0, trgsize, stream[i]);
    hipMemsetAsync (dtv[i], 0, trgsize, stream[i]);
    hipMemsetAsync (dtw[i], 0, trgsize, stream[i]);
    // now we need to be careful to point to the part of the source arrays that hold
    //   just this GPUs set of target particles
    dtx[i] = dsx[i] + i*ntargperstrm;
    dty[i] = dsy[i] + i*ntargperstrm;
    dtz[i] = dsz[i] + i*ntargperstrm;
    dtr[i] = dsr[i] + i*ntargperstrm;

    // check
    auto memerr = hipGetLastError();
    if (memerr != hipSuccess) {
      fprintf(stderr, "Failed to upload data (other): %s!\n", hipGetErrorString(memerr));
      exit(EXIT_FAILURE);
    }
  }

    const dim3 blocksz(THREADS_PER_BLOCK, 1, 1);
    const dim3 gridsz(ntargperstrm/THREADS_PER_BLOCK, nsrcblocks, 1);

  for (int32_t i=0; i<nstreams; ++i) {
    // launch the kernel
    hipSetDevice(i);
    ngrav_3d_nograds_gpu<<<gridsz,blocksz,0,stream[i]>>>(nsrcpad, dsx[i],dsy[i],dsz[i],dss[i],dsr[i],
                                               0,dtx[i],dty[i],dtz[i],dtr[i],dtu[i],dtv[i],dtw[i]);

    // check
    auto err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Failed to launch kernel (%d): %s!\n", i, hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
  }

  for (int32_t i=0; i<nstreams; ++i) {
    // pull data back down
    hipMemcpyAsync (htu.data() + i*ntargperstrm, dtu[i], trgsize, hipMemcpyDeviceToHost, stream[i]);
    hipMemcpyAsync (htv.data() + i*ntargperstrm, dtv[i], trgsize, hipMemcpyDeviceToHost, stream[i]);
    hipMemcpyAsync (htw.data() + i*ntargperstrm, dtw[i], trgsize, hipMemcpyDeviceToHost, stream[i]);
  }

  // join streams
  for (int32_t i=0; i<nstreams; ++i) {
    hipStreamSynchronize(stream[i]);
  }
  //cudaDeviceSynchronize();

  // time and report
  auto end = std::chrono::system_clock::now();
  std::chrono::duration<double> elapsed_seconds = end-start;
  double time = elapsed_seconds.count();
  printf( "  device total time( %g s ) and flops( %g GFlop/s )\n", time, 1.e-9 * (double)npart*(6+22*(double)npart)/time);
  for (int i=0; i<4; ++i) {
    printf( "    part %d acc %g %g %g)\n", i, htu[i], htv[i], htw[i]);
  }

  // free resources, after timer
  for (int32_t i=0; i<nstreams; ++i) {
    hipFree(dsx[i]);
    hipFree(dsy[i]);
    hipFree(dsz[i]);
    hipFree(dss[i]);
    hipFree(dsr[i]);
    hipFree(dtu[i]);
    hipFree(dtv[i]);
    hipFree(dtw[i]);
    hipStreamDestroy(stream[i]);
  }

  // compare results
  if (compare) {
  FLOAT errsum = 0.0;
  FLOAT errmax = 0.0;
  for (int32_t i=0; i<npart; ++i) {
    const FLOAT thiserr = std::pow(htu[i]-htu_cpu[i], 2) + std::pow(htv[i]-htv_cpu[i], 2);
    errsum += thiserr;
    if ((FLOAT)std::sqrt(thiserr) > errmax) {
      errmax = (FLOAT)std::sqrt(thiserr);
      //printf( "    err at %d is %g\n", i, errmax);
    }
  }
  printf( "  total host-device error ( %g ) max error ( %g )\n", std::sqrt(errsum/npart), errmax);
  }
}

