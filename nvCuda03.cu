/*
 * nvortexCuda.cpp
 *
 * (c)2022 Mark J. Stock <markjstock@gmail.com>
 *
 * v0.3  use atomics to expose more concurrency
 */

#include <vector>
#include <random>
#include <chrono>

#include <hip/hip_runtime.h>


// compute using float or double
#define FLOAT float

// threads per block (hard coded)
#define THREADS_PER_BLOCK 128

// GPU count limit
#define MAX_GPUS 8

// -------------------------
// compute kernel - GPU
__global__ void nvortex_2d_nograds_gpu(
    const int32_t nSrc,
    const FLOAT* const sx,
    const FLOAT* const sy,
    const FLOAT* const ss,
    const FLOAT* const sr,
    const int32_t tOffset,
    const FLOAT* const tx,
    const FLOAT* const ty,
    const FLOAT* const tr,
    FLOAT* const tu,
    FLOAT* const tv) {

  // local "thread" id - this is the target particle
  const int32_t i = tOffset + blockIdx.x*THREADS_PER_BLOCK + threadIdx.x;

  // load sources into shared memory (or not)
  __shared__ FLOAT s_sx[THREADS_PER_BLOCK];
  __shared__ FLOAT s_sy[THREADS_PER_BLOCK];
  __shared__ FLOAT s_ss[THREADS_PER_BLOCK];
  __shared__ FLOAT s_sr[THREADS_PER_BLOCK];

  // velocity accumulators for target point
  FLOAT locu = 0.0f;
  FLOAT locv = 0.0f;

  // which sources do we iterate over?
  const int32_t jcount = nSrc / gridDim.y;
  const int32_t jstart = blockIdx.y * jcount;

  for (int32_t b=0; b<jcount/THREADS_PER_BLOCK; ++b) {
    __syncthreads();

    const int32_t gidx = jstart + b*THREADS_PER_BLOCK + threadIdx.x;
    s_sx[threadIdx.x] = sx[gidx];
    s_sy[threadIdx.x] = sy[gidx];
    s_ss[threadIdx.x] = ss[gidx];
    s_sr[threadIdx.x] = sr[gidx];
    __syncthreads();

    // loop over all source points
    for (int32_t j=0; j<THREADS_PER_BLOCK; ++j) {
      FLOAT dx = s_sx[j] - tx[i];
      FLOAT dy = s_sy[j] - ty[i];
      FLOAT distsq = dx*dx + dy*dy + s_sr[j]*s_sr[j] + tr[i]*tr[i];
      // consider __fdividef(x, y) ?
      FLOAT factor = s_ss[j] / distsq;
      locu += dy * factor;
      locv -= dx * factor;
    }
  }

  // save into device view
  //tu[i] = locu / (2.0f*3.1415926536f);
  //tv[i] = locv / (2.0f*3.1415926536f);
  // use atomics
  atomicAdd(&tu[i], locu / (2.0f*3.1415926536f));
  atomicAdd(&tv[i], locv / (2.0f*3.1415926536f));

  return;
}

// -------------------------
// compute kernel - CPU
__host__ void nvortex_2d_nograds_cpu(
    const int32_t nSrc,
    const FLOAT* const __restrict__ sx,
    const FLOAT* const __restrict__ sy,
    const FLOAT* const __restrict__ ss,
    const FLOAT* const __restrict__ sr,
    const FLOAT tx,
    const FLOAT ty,
    const FLOAT tr,
    FLOAT* const tu,
    FLOAT* const tv) {

  // velocity accumulators for target point
  FLOAT locu = 0.0f;
  FLOAT locv = 0.0f;

  // loop over all source points
  #pragma omp simd reduction(+:locu,locv)
  for (int32_t j=0; j<nSrc; ++j) {
    FLOAT dx = sx[j] - tx;
    FLOAT dy = sy[j] - ty;
    FLOAT distsq = dx*dx + dy*dy + sr[j]*sr[j] + tr*tr;
    FLOAT factor = ss[j] / distsq;
    locu += dy * factor;
    locv -= dx * factor;
  }

  // save into device view
  // use atomics?!?
  *tu = locu / (2.0f*3.1415926536f);
  *tv = locv / (2.0f*3.1415926536f);

  return;
}

// not really alignment, just minimum block sizes
__host__ int32_t buffer(const int32_t _n, const int32_t _align) {
  // 63,64 returns 1; 64,64 returns 1; 65,64 returns 2
  return _align*(1+(_n-1)/_align);
}

int main(int argc, char **argv) {

  // number of particles/points
  const int32_t npart = 400000;
  printf( "performing 2D vortex Biot-Savart on %d points\n", npart);

  // number of GPUs present
  int32_t ngpus = 1;
  hipGetDeviceCount(&ngpus);
  //ngpus = 1;	// Force 1 GPU
  // number of cuda streams to break work into
  int32_t nstreams = std::min(MAX_GPUS, ngpus);
  printf( "  ngpus ( %d )  and nstreams ( %d )\n", ngpus, nstreams);

  // we parallelize targets over GPUs/streams
  const int32_t ntargperstrm = buffer(npart/nstreams, THREADS_PER_BLOCK*nstreams);
  const int32_t ntargpad = ntargperstrm * nstreams;
  printf( "  ntargperstrm ( %d )  and ntargpad ( %d )\n", ntargperstrm, ntargpad);

  // and on each GPU, we parallelize over THREADS_PER_BLOCK targets and nsrcblocks source blocks
  // number of blocks source-wise (break summations over sources into this many chunks)
  const int32_t nsrcblocks = 16;

  // set stream sizes
  const int32_t nsrcpad = buffer(npart, THREADS_PER_BLOCK*nsrcblocks);
  const int32_t nsrcperblock = nsrcpad / nsrcblocks;
  printf( "  nsrcperblock ( %d )  and nsrcpad ( %d )\n", nsrcperblock, nsrcpad);

  // define the host arrays (for now, sources and targets are the same)
  const int32_t npad = std::max(ntargpad,nsrcpad);
  std::vector<FLOAT> hsx(npad), hsy(npad), hss(npad), hsr(npad), htu(npad), htv(npad);
  const FLOAT thisstrmag = 1.0 / std::sqrt(npart);
  const FLOAT thisrad    = (2./3.) / std::sqrt(npart);
  //std::random_device dev;
  //std::mt19937 rng(dev());
  std::mt19937 rng(1234);
  std::uniform_real_distribution<FLOAT> xrand(0.0,1.0);
  for (int32_t i = 0; i < npart; ++i)    hsx[i] = xrand(rng);
  for (int32_t i = npart; i < npad; ++i) hsx[i] = 0.0;
  for (int32_t i = 0; i < npart; ++i)    hsy[i] = xrand(rng);
  for (int32_t i = npart; i < npad; ++i) hsy[i] = 0.0;
  for (int32_t i = 0; i < npart; ++i)    hss[i] = thisstrmag * (2.0*xrand(rng)-1.0);
  for (int32_t i = npart; i < npad; ++i) hss[i] = 0.0;
  for (int32_t i = 0; i < npart; ++i)    hsr[i] = thisrad;
  for (int32_t i = npart; i < npad; ++i) hsr[i] = thisrad;
  for (int32_t i = 0; i < npad; ++i)     htu[i] = 0.0;
  for (int32_t i = 0; i < npad; ++i)     htv[i] = 0.0;

  // -------------------------
  // do a CPU version

  auto start = std::chrono::system_clock::now();

  #pragma omp parallel for
  for (int32_t i=0; i<npart; ++i) {
    nvortex_2d_nograds_cpu(npart, hsx.data(),hsy.data(),hss.data(),hsr.data(), hsx[i],hsy[i],hsr[i], &htu[i],&htv[i]);
  }

  auto end = std::chrono::system_clock::now();
  std::chrono::duration<double> elapsed_seconds = end-start;
  double time = elapsed_seconds.count();

  printf( "  host total time( %g s ) and flops( %g GFlop/s )\n", time, 1.e-9 * (double)npart*(4+14*(double)npart)/time);
  printf( "    results ( %g %g %g %g %g %g)\n", htu[0], htv[0], htu[1], htv[1], htu[npart-1], htv[npart-1]);

  // copy the results into temp vectors
  std::vector<FLOAT> htu_cpu(htu);
  std::vector<FLOAT> htv_cpu(htv);

  // -------------------------
  // do the GPU version

  // set device pointers, too
  FLOAT *dsx[MAX_GPUS], *dsy[MAX_GPUS], *dss[MAX_GPUS], *dsr[MAX_GPUS];
  FLOAT *dtx[MAX_GPUS], *dty[MAX_GPUS], *dtr[MAX_GPUS];
  FLOAT *dtu[MAX_GPUS], *dtv[MAX_GPUS];
  hipStream_t stream[MAX_GPUS];

  // allocate space for all sources, part of targets
  const int32_t srcsize = nsrcpad*sizeof(FLOAT);
  const int32_t trgsize = ntargperstrm*sizeof(FLOAT);
  for (int32_t i=0; i<nstreams; ++i) {
    hipSetDevice(i);
    hipStreamCreate(&stream[i]);

    hipMalloc (&dsx[i], srcsize);
    hipMalloc (&dsy[i], srcsize);
    hipMalloc (&dss[i], srcsize);
    hipMalloc (&dsr[i], srcsize);
    hipMalloc (&dtu[i], trgsize);
    hipMalloc (&dtv[i], trgsize);
  }

  // to be fair, we start timer after allocation but before transfer
  start = std::chrono::system_clock::now();

  // now perform the data movement and setting
  for (int32_t i=0; i<nstreams; ++i) {

    hipSetDevice(i);

    // move the data
    hipMemcpyAsync (dsx[i], hsx.data(), srcsize, hipMemcpyHostToDevice, stream[i]);
    hipMemcpyAsync (dsy[i], hsy.data(), srcsize, hipMemcpyHostToDevice, stream[i]);
    hipMemcpyAsync (dss[i], hss.data(), srcsize, hipMemcpyHostToDevice, stream[i]);
    hipMemcpyAsync (dsr[i], hsr.data(), srcsize, hipMemcpyHostToDevice, stream[i]);
    hipMemsetAsync (dtu[i], 0, trgsize, stream[i]);
    hipMemsetAsync (dtv[i], 0, trgsize, stream[i]);
    // now we need to be careful to point to the part of the source arrays that hold
    //   just this GPUs set of target particles
    dtx[i] = dsx[i] + i*ntargperstrm;
    dty[i] = dsy[i] + i*ntargperstrm;
    dtr[i] = dsr[i] + i*ntargperstrm;

    // check
    auto memerr = hipGetLastError();
    if (memerr != hipSuccess) {
      fprintf(stderr, "Failed to upload data (other): %s!\n", hipGetErrorString(memerr));
      exit(EXIT_FAILURE);
    }
  }

    const dim3 blocksz(THREADS_PER_BLOCK, 1, 1);
    const dim3 gridsz(ntargperstrm/THREADS_PER_BLOCK, nsrcblocks, 1);

  for (int32_t i=0; i<nstreams; ++i) {
    // launch the kernel
    hipSetDevice(i);
    nvortex_2d_nograds_gpu<<<gridsz,blocksz,0,stream[i]>>>(nsrcpad, dsx[i],dsy[i],dss[i],dsr[i],
                                               0,dtx[i],dty[i],dtr[i],dtu[i],dtv[i]);

    // check
    auto err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Failed to launch kernel (%d): %s!\n", i, hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
  }

  for (int32_t i=0; i<nstreams; ++i) {
    // pull data back down
    hipMemcpyAsync (htu.data() + i*ntargperstrm, dtu[i], trgsize, hipMemcpyDeviceToHost, stream[i]);
    hipMemcpyAsync (htv.data() + i*ntargperstrm, dtv[i], trgsize, hipMemcpyDeviceToHost, stream[i]);
  }

  // join streams
  for (int32_t i=0; i<nstreams; ++i) {
    hipStreamSynchronize(stream[i]);
  }
  //cudaDeviceSynchronize();

  // time and report
  end = std::chrono::system_clock::now();
  elapsed_seconds = end-start;
  time = elapsed_seconds.count();
  printf( "  device total time( %g s ) and flops( %g GFlop/s )\n", time, 1.e-9 * (double)npart*(4+14*(double)npart)/time);
  printf( "    results ( %g %g %g %g %g %g)\n", htu[0], htv[0], htu[1], htv[1], htu[npart-1], htv[npart-1]);

  // free resources, after timer
  for (int32_t i=0; i<nstreams; ++i) {
    hipFree(dsx[i]);
    hipFree(dsy[i]);
    hipFree(dss[i]);
    hipFree(dsr[i]);
    hipFree(dtu[i]);
    hipFree(dtv[i]);
    hipStreamDestroy(stream[i]);
  }

  // compare results
  FLOAT errsum = 0.0;
  FLOAT errmax = 0.0;
  for (int32_t i=0; i<npart; ++i) {
    const FLOAT thiserr = std::pow(htu[i]-htu_cpu[i], 2) + std::pow(htv[i]-htv_cpu[i], 2);
    errsum += thiserr;
    if ((FLOAT)std::sqrt(thiserr) > errmax) {
      errmax = (FLOAT)std::sqrt(thiserr);
      //printf( "    err at %d is %g\n", i, errmax);
    }
  }
  printf( "  total host-device error ( %g ) max error ( %g )\n", std::sqrt(errsum/npart), errmax);
}

